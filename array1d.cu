
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    float A[] = {1,2,3};
    float B[] = {2,3,4};
    float C[] = {0,0,0};

    float *a, *b, *c;
    hipMalloc(&a, 3*sizeof(float));
    hipMalloc(&b, 3*sizeof(float));
    hipMalloc(&c, 3*sizeof(float));

    hipMemcpy(a, A, 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, B, 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, C, 3*sizeof(float), hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, 3>>>(a, b, c);

    hipMemcpy(C, c, 3*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << C[0] << " " << C[1] << " " << C[2] << std::endl;

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}


/* reference */ 
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#programming-model
// https://qiita.com/wazakkyd/items/8a5694e7a001465b6025
