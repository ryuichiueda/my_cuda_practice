
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void MatAdd(float *A, float *B, float *C)
{
    int block_idx = blockIdx.x*blockDim.x*blockDim.y;
    int i = threadIdx.x + blockDim.x * threadIdx.y + block_idx;
    for(int k=0;k<1024;k++)
        C[i] += A[i]*3.14 + B[i]/3.14;
}

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    std::cout << N << std::endl;

    float A[N*N], B[N*N], C[N*N];
    for(int j=0;j<N;j++){
    	for(int i=0;i<N;i++){
	   A[i + j*N] = i;
	   B[i + j*N] = j;
	   C[i + j*N] = 0.0;
	}
    }

    auto start = std::chrono::system_clock::now();

    float *a, *b, *c;
    hipMalloc(&a, N*N*sizeof(float));
    hipMalloc(&b, N*N*sizeof(float));
    hipMalloc(&c, N*N*sizeof(float));

    hipMemcpy(a, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, C, N*N*sizeof(float), hipMemcpyHostToDevice);

    MatAdd<<<N, N>>>(a, b, c);

    hipMemcpy(C, c, N*N*sizeof(float), hipMemcpyDeviceToHost);
    auto end = std::chrono::system_clock::now();
    auto dur = end - start;
    std::cerr << std::chrono::duration_cast<std::chrono::milliseconds>(dur).count() << "msec" << std::endl;

    std::cout << "C" << std::endl;
    for(int j=N-1;j<N;j++){
    	for(int i=0;i<N;i++){
		std::cout << C[i + j*N] << ' ';
	}
	std::cout << std::endl;
    }


    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}


/* reference */ 
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#programming-model
// https://qiita.com/wazakkyd/items/8a5694e7a001465b6025
